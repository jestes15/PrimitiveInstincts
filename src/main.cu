#include "cuda_mem_ctrl.h"
#include "kernels.h"

int main()
{
    hipInit(0);

    uint8_t *src = nullptr, *dst = nullptr;
    cuda_mem_ctrl::cuda_cpy(src, dst, 0, hipMemcpyDefault);
}