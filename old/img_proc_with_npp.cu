#include <iostream>

#include <opencv2/opencv.hpp>


int main()
{
    hipSetDevice(0);

    std::cout << "##########################################################" << std::endl;
    std::cout << "# Image Processing Using NVIDIA Performance Primitives   #" << std::endl;
    std::cout << "#\tCreated by Joshua Estes                          #" << std::endl;
    std::cout << "##########################################################" << std::endl;

    uint8_t *device_source_image = nullptr;
    uint8_t *device_intermediate_storage = nullptr;
    uint8_t *device_dst_image = nullptr;
    uint8_t *host_int = nullptr;
    uint8_t *final_image = nullptr;

    // Load the image
    cv::Mat image =

    int width = image.cols;
    int height = image.rows;

    hipMalloc((void **)&device_source_image, width * height * image_sizes::RGB24_bytes_per_pixel);
    hipMalloc((void **)&device_intermediate_storage, width * height * image_sizes::CbYCr422_bytes_per_pixel);
    hipMalloc((void **)&device_dst_image, width * height * image_sizes::RGB24_bytes_per_pixel);

    hipHostMalloc((void **)&final_image, width * height * image_sizes::RGB24_bytes_per_pixel);
    hipHostMalloc((void **)&host_int, width * height * image_sizes::CbYCr422_bytes_per_pixel);

    // Check if the image was loaded successfully
    if (image.empty())
    {
        std::cout << "Could not open or find the image!" << std::endl;
        return -1;
    }

    for (int i = 0; i < 6; i++)
        printf("data[%d] - > %d\n", i, image.data[i]);

    // Copy the CbYCr422 data to device memory
    hipMemcpy(device_source_image,
               image.data,
               width * height * image_sizes::RGB24_bytes_per_pixel,
               hipMemcpyHostToDevice);

    NppiSize roiSize = {.width = width, .height = height};
    printf("nppi func ret: %d\n",
           nppiRGBToCbYCr422_8u_C3C2R(device_source_image,
                                          width * image_sizes::RGB24_bytes_per_pixel,
                                          device_intermediate_storage,
                                          width * image_sizes::CbYCr422_bytes_per_pixel,
                                          roiSize));

    // nppiRGB24ToCbYCr422(device_source_image, device_intermediate_storage, width, height);

    hipMemcpy(host_int,
               device_intermediate_storage,
               width * height * image_sizes::CbYCr422_bytes_per_pixel,
               hipMemcpyDeviceToHost);

    for (int i = 0; i < 6; i++)
        printf("src[%d] - > %d\n", i, host_int[i]);

    convert_CbYCr_To_BGR24(device_intermediate_storage, device_dst_image, height, width);

    // Copy the result back to host memory
    hipMemcpy(final_image,
               device_dst_image,
               width * height * image_sizes::RGB24_bytes_per_pixel,
               hipMemcpyDeviceToHost);

    // Create a Mat for the RGB image
    // cv::Mat rgbImage(height, width, CV_8UC3, final_image);
    // cv::imshow("images_1920x1080/image_23.jpg pre-process", image);
    // cv::imshow("images_1920x1080/image_23.jpg post-process", rgbImage);

    hipFree(device_source_image);
    hipFree(device_intermediate_storage);
    hipFree(device_dst_image);
    hipHostFree(final_image);
    hipHostFree(host_int);

    hipDeviceReset();

    // cv::waitKey(0);          // Wait for a key press
    // cv::destroyAllWindows(); // Close all OpenCV windows
    return 0;
}